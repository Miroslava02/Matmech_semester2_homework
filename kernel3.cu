#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""
#include <ctime>
#include <stdio.h>
#include<iostream>
using namespace std;

#define N 70000//количество элементов массива
// Ядро
#define K 100;

__global__ void add(int* a, int* b, int* c)
{
	//вычисление индекса элемента
	int tid = threadIdx.x;
	//проверка на выход за пределы массива
	if (tid > N - 1) return;
	//поэлементное сложение массивов
	c[tid] = a[tid] + b[tid];
}
int main() {
	double start_time = clock();
	// выделение памяти под массивы на CPU
	int host_a[N], host_b[N], host_c[N];
	// выделение памяти под массивы для копирования
	// на GPU
	int* dev_a, * dev_b, * dev_c;
	// заполнение массивов
	for (int i = 0; i < N; i++)
	{
		host_a[i] = 23;
		host_b[i] = 23;
	}
	// выделение памяти под массивы на GPU
	hipMalloc((void**)&dev_a, N * sizeof(int)  );
	hipMalloc((void**)&dev_b, N * sizeof(int));
	hipMalloc((void**)&dev_c, N * sizeof(int)  );
	// копирование данных в память GPU
	hipMemcpy(dev_a, host_a, N * sizeof(int)   , 
		hipMemcpyHostToDevice);
		hipMemcpy(dev_b, host_b, N * sizeof(int)  , 
		hipMemcpyHostToDevice);
	// вызов ядра
		add << <1, N  >> > (dev_a, dev_b, dev_c);
	// получаем результат расчета
	hipMemcpy(host_c, dev_c, N * sizeof(int) , 
		hipMemcpyDeviceToHost);
	// вывод результатов
	//for (int i = 0; i < N; i++)
	//{
		//printf("%d + %d = %d\n", host_a[i],
			//host_b[i], host_c[i]);
	//}
	// освобождение памяти
	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);
	double end_time = clock(); // конечное время
	double search_time = end_time - start_time; // искомое время
	cout << N << endl;
	cout << search_time;
	return 0;
}
