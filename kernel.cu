#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""
#include <ctime>
#include <stdio.h>
#include<iostream>
using namespace std;

#define N  10000//количество элементов массива
// Ядро

#define M 7  //тво массивов

__global__ void add(int* a, int* b, int* c)
{
	//вычисление индекса элемента
	int tid = threadIdx.x;
	//проверка на выход за пределы массива
	if (tid > N - 1) return;
	//поэлементное сложение массивов
	c[tid] = a[tid] + b[tid];
}
int main() {
	double start_time = clock();
	// выделение памяти под массивы на CPU
	int host_a[M][N];
	int host_b[M][N];
	int host_c[M][N];
	// выделение памяти под массивы для копирования
	// на GPU
	int* dev_a, * dev_b, * dev_c;
	// заполнение массивов
	for (int i = 0; i < M; i++) {
		for (int j = 0; j < N; j++) 
		{
			host_a[i][j] = 23;
			host_b[i][j] = 23;
		}
	}
	// выделение памяти под массивы на GPU
	hipMalloc((void**)&dev_a, N * M * sizeof(int));
	hipMalloc((void**)&dev_b, N * M * sizeof(int));
	hipMalloc((void**)&dev_c, N * M * sizeof(int));
	// копирование данных в память GPU
	hipMemcpy(dev_a, host_a, N *M*sizeof(int),
		hipMemcpyHostToDevice);
	hipMemcpy(dev_b, host_b, N * sizeof(int)*M,
		hipMemcpyHostToDevice);
	// вызов ядра
	add << <1, N*M>>>(dev_a, dev_b, dev_c);
	// получаем результат расчета
	hipMemcpy(host_c, dev_c, N * sizeof(int)*M,
		hipMemcpyDeviceToHost);
	// вывод результатов
	//for (int i = 0; i < N; i++)
	//{
		//printf("%d + %d = %d\n", host_a[i],
			//host_b[i], host_c[i]);
	//}
	// освобождение памяти
	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);
	double end_time = clock(); // конечное время
	double search_time = end_time - start_time; // искомое время
	cout << N << endl << M << endl;
	cout << search_time;
	return 0;
}
